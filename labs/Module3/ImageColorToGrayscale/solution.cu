#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


#define BLOCK_SIZE  (16)

__global__ void ConvertToGrayScale(float * colorImage, float * grayImage, int width, int height, int channels)
{
  int x = threadIdx.x + blockIdx.x * blockDim.x; // Column
  int y = threadIdx.y + blockIdx.y * blockDim.y; // Row

  if ((x < width) && (y < height))
  {
    int grayOffset = y * width + x;
    int colorOffset = grayOffset * channels;

	float red = colorImage[colorOffset];
	float green = colorImage[colorOffset + 1];
	float blue = colorImage[colorOffset + 2];
	float grayValue = 0.21f * red + 0.71f * green + 0.07 * blue;

	if ((x == 0) && (y == 0))
	{
		printf("\n\n[%d, %d]:rgb (%f, %f, %f); gray (%f)\n\n", x, y, red, green, blue, grayValue);
	}
    grayImage[grayOffset] = grayValue;
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  
  dim3 DimGrid((imageWidth-1)/BLOCK_SIZE + 1, (imageHeight-1)/BLOCK_SIZE + 1, 1);
  dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  ConvertToGrayScale<<<DimGrid,DimBlock>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight, imageChannels);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
